#include "hip/hip_runtime.h"
// BSD 3-Clause License
//
// Copyright (c) 2025, Yafei Ou and Mahdi Tavakoli
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "gpu_standard_mpm_solver.cuh"
#include "standard_mpm_algorithm.h"
#include "constants.h"
#include "check_cuda.cuh"

namespace crmpm
{
    GpuStandardMpmSolver::GpuStandardMpmSolver(int numParticles, float cellSize, Bounds3 gridBound) : MpmSolverBase()
    {
        // Pre-computed values for simulation
        mNumMaxParticles = numParticles;

        mGridBound = gridBound;
        mCellSize = cellSize;
        mInvCellSize = 1 / cellSize;
        Vec3f gridSize = mGridBound.maximum - mGridBound.minimum;
        mNumNodesPerDim = (gridSize / cellSize + 1.0f).cast<int>();
        mNumNodes = mNumNodesPerDim.x() * mNumNodesPerDim.y() * mNumNodesPerDim.z();
        mGridVolume = cellSize * cellSize * cellSize;
    }

    void GpuStandardMpmSolver::initialize()
    {
        dmParticlePositionMass = mParticleData->positionMass;
        dmParticleVelocity = mParticleData->velocity;
        dmParticleMaterialProperties0 = mParticleMaterialData->params0;
        dmParticleMaterialTypes = mParticleMaterialData->type;

        // GPU particle data
        CR_CHECK_CUDA(hipMalloc<float>(&dmParticleInitialVolume, mNumMaxParticles * sizeof(float)));
        CR_CHECK_CUDA(hipMalloc<float4>(&dmParticleGradientDeformationTensorColumn0, mNumMaxParticles * sizeof(float4)));
        CR_CHECK_CUDA(hipMalloc<float4>(&dmParticleGradientDeformationTensorColumn1, mNumMaxParticles * sizeof(float4)));
        CR_CHECK_CUDA(hipMalloc<float4>(&dmParticleGradientDeformationTensorColumn2, mNumMaxParticles * sizeof(float4)));

        // GPU node data
        CR_CHECK_CUDA(hipMalloc<float4>(&dmNodeMomentumVelocityMass, mNumNodes * sizeof(float4)));
        CR_CHECK_CUDA(hipMalloc<Vec3f>(&dmNodeForce, mNumNodes * sizeof(Vec3f)));

        // Reset all GPU data to zero
        CR_CHECK_CUDA(hipMemset(dmParticleInitialVolume, 0, mNumMaxParticles * sizeof(float)));
        CR_CHECK_CUDA(hipMemset(dmParticleGradientDeformationTensorColumn0, 0, mNumMaxParticles * sizeof(float4)));
        CR_CHECK_CUDA(hipMemset(dmParticleGradientDeformationTensorColumn1, 0, mNumMaxParticles * sizeof(float4)));
        CR_CHECK_CUDA(hipMemset(dmParticleGradientDeformationTensorColumn2, 0, mNumMaxParticles * sizeof(float4)));

        CR_CHECK_CUDA(hipMemset(dmNodeMomentumVelocityMass, 0, mNumNodes * sizeof(float4)));
        CR_CHECK_CUDA(hipMemset(dmNodeForce, 0, mNumNodes * sizeof(Vec3f)));
    }

    void GpuStandardMpmSolver::computeInitialData(unsigned int numParticlesToCompute,
                                                         const unsigned int *CR_RESTRICT indices)
    {
        // Initial P2G and G2P for particle volume
        int blockSize = 128;
        dim3 block(blockSize);
        dim3 grid((numParticlesToCompute + block.x - 1) / block.x);
        standardMpmComputeInitialGridMassKernel<<<grid, block>>>(numParticlesToCompute, mGridBound.minimum, mInvCellSize, mNumNodesPerDim, mGridVolume, dmParticlePositionMass,
                                                                 dmParticleGradientDeformationTensorColumn0, dmParticleGradientDeformationTensorColumn1,
                                                                 dmParticleGradientDeformationTensorColumn2, dmNodeMomentumVelocityMass);
        standardMpmComputeInitialVolumeKernel<<<grid, block>>>(numParticlesToCompute, mGridBound.minimum, mInvCellSize, mNumNodesPerDim, mGridVolume, dmParticlePositionMass,
                                                               dmNodeMomentumVelocityMass,
                                                               dmParticleGradientDeformationTensorColumn0, dmParticleGradientDeformationTensorColumn1,
                                                               dmParticleGradientDeformationTensorColumn2, dmParticleInitialVolume);
        CR_CHECK_CUDA(hipDeviceSynchronize());
    }

    void GpuStandardMpmSolver::resetGrid()
    {
        CR_CHECK_CUDA(hipMemset(dmNodeMomentumVelocityMass, 0, mNumNodes * sizeof(float4)));
        CR_CHECK_CUDA(hipMemset(dmNodeForce, 0, mNumNodes * sizeof(Vec3f)));
    }

    void GpuStandardMpmSolver::particleToGrid()
    {
        int blockSize = 128;
        dim3 block(blockSize);
        dim3 grid((mNumActiveParticles + block.x - 1) / block.x);
        standardMpmParticleToGridKernel<<<grid, block>>>(mNumActiveParticles, mGravity, mGridBound.minimum, mInvCellSize, mNumNodesPerDim,
                                                         dmParticlePositionMass, dmParticleVelocity, dmParticleInitialVolume,
                                                         dmParticleGradientDeformationTensorColumn0, dmParticleGradientDeformationTensorColumn1,
                                                         dmParticleGradientDeformationTensorColumn2,
                                                         dmParticleMaterialProperties0, dmParticleMaterialTypes,
                                                         dmNodeMomentumVelocityMass, dmNodeForce);
    }

    void GpuStandardMpmSolver::updateGrid()
    {
        int blockSize = 128;
        dim3 block(blockSize);
        dim3 grid((mNumNodes + block.x - 1) / block.x);
        standardMpmUpdateGridKernel<<<grid, block>>>(
            mNumNodes,
            mGridBound.minimum,
            mCellSize,
            mNumNodesPerDim,
            mIntegrationStepSize,
            dmNodeForce,
            mNumShapes,
            mShapeIds,
            *mShapeData,
            *mGeometryData,
            *mGeometrySdfData,
            dmNodeMomentumVelocityMass);
    }

    void GpuStandardMpmSolver::gridToParticle()
    {
        int blockSize = 128;
        dim3 block(blockSize);
        dim3 grid((mNumActiveParticles + block.x - 1) / block.x);
        standardMpmGridToParticleKernel<<<grid, block>>>(
            mNumActiveParticles,
            mGridBound.minimum,
            mGridBound.maximum,
            mInvCellSize,
            mNumNodesPerDim,
            mIntegrationStepSize,
            dmNodeMomentumVelocityMass,
            mNumShapes,
            mShapeIds, // The data is on GPU
            *mShapeData,
            *mGeometryData,
            *mGeometrySdfData,
            dmParticlePositionMass,
            dmParticleVelocity,
            dmParticleGradientDeformationTensorColumn0,
            dmParticleGradientDeformationTensorColumn1,
            dmParticleGradientDeformationTensorColumn2);
    }

    float GpuStandardMpmSolver::step()
    {
        resetGrid();
        particleToGrid();
        updateGrid();
        gridToParticle();
        return mIntegrationStepSize;
    }

    void GpuStandardMpmSolver::fetchResults()
    {
        CR_CHECK_CUDA(hipDeviceSynchronize());
    }

    void GpuStandardMpmSolver::_release()
    {
        // Device
        CR_CHECK_CUDA(hipFree(dmParticleInitialVolume));
        CR_CHECK_CUDA(hipFree(dmParticleGradientDeformationTensorColumn0));
        CR_CHECK_CUDA(hipFree(dmParticleGradientDeformationTensorColumn1));
        CR_CHECK_CUDA(hipFree(dmParticleGradientDeformationTensorColumn2));

        CR_CHECK_CUDA(hipFree(dmNodeMomentumVelocityMass));
        CR_CHECK_CUDA(hipFree(dmNodeForce));
    }

    ParticleData &GpuStandardMpmSolver::getParticleData()
    {
        return *mParticleData;
    }

    CR_CUDA_GLOBAL void standardMpmComputeInitialGridMassKernel(
        const int numParticles,
        const Vec3f gridBoundMin,
        const float invCellSize,
        const Vec3i numNodesPerDim,
        const float gridVolume,
        const float4 *CR_RESTRICT particlePositionMass,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn0,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn1,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn2,
        float4 *CR_RESTRICT nodeMomentumVelocityMass)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        // Set deformation tensor to identity
        particleGradientDeformationTensorColumn0[idx].x = 1;
        particleGradientDeformationTensorColumn1[idx].y = 1;
        particleGradientDeformationTensorColumn2[idx].z = 1;

        standardMpmComputeInitialGridMass<true>(
            gridBoundMin,
            invCellSize,
            numNodesPerDim,
            gridVolume,
            particlePositionMass[idx],
            nodeMomentumVelocityMass);
    }

    CR_CUDA_GLOBAL void standardMpmComputeInitialVolumeKernel(
        const int numParticles,
        const Vec3f gridBoundMin,
        const float invCellSize,
        const Vec3i numNodesPerDim,
        const float gridVolume,
        const float4 *CR_RESTRICT particlePositionMass,
        const float4 *CR_RESTRICT nodeMomentumVelocityMass,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn0,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn1,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn2,
        float *CR_RESTRICT particleInitialVolume)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        // Set deformation tensor to identity
        particleGradientDeformationTensorColumn0[idx].x = 1;
        particleGradientDeformationTensorColumn1[idx].y = 1;
        particleGradientDeformationTensorColumn2[idx].z = 1;

        standardMpmComputeInitialVolume(
            gridBoundMin,
            invCellSize,
            numNodesPerDim,
            gridVolume,
            particlePositionMass[idx],
            nodeMomentumVelocityMass,
            particleInitialVolume[idx]);
    }

    /**
     * @brief Standard MPM P2G kernel
     */
    CR_CUDA_GLOBAL void standardMpmParticleToGridKernel(
        const int numParticles,
        const Vec3f gravity,
        const Vec3f gridBoundMin,
        const float invCellSize,
        const Vec3i numNodesPerDim,
        const float4 *CR_RESTRICT particlePositionMass,
        const Vec3f *CR_RESTRICT particleVelocity,
        const float *CR_RESTRICT particleInitialVolume,
        const float4 *CR_RESTRICT particleGradientDeformationTensorColumn0,
        const float4 *CR_RESTRICT particleGradientDeformationTensorColumn1,
        const float4 *CR_RESTRICT particleGradientDeformationTensorColumn2,
        const float4 *CR_RESTRICT particleMaterialProperties0,
        const ParticleMaterialType *CR_RESTRICT particleMaterialTypes,
        float4 *CR_RESTRICT nodeMomentumVelocityMass,
        Vec3f *CR_RESTRICT nodeForce)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        // Reconstruct tensors
        Mat3x3f particleDeformationTensor(particleGradientDeformationTensorColumn0[idx],
                                          particleGradientDeformationTensorColumn1[idx],
                                          particleGradientDeformationTensorColumn2[idx]);

        standardMpmParticleToGrid<true>(
            gravity,
            gridBoundMin,
            invCellSize,
            numNodesPerDim,
            particlePositionMass[idx],
            particleVelocity[idx],
            particleInitialVolume[idx],
            particleDeformationTensor,
            particleMaterialProperties0[idx],
            particleMaterialTypes[idx],
            nodeMomentumVelocityMass,
            nodeForce);
    }

    CR_CUDA_GLOBAL void standardMpmUpdateGridKernel(
        const int numNodes,
        const Vec3f gridBoundMin,
        const float cellSize,
        const Vec3i numNodesPerDim,
        const float integrationStepSize,
        const Vec3f *CR_RESTRICT nodeForce,
        const int numShapes,
        const int *CR_RESTRICT shapeIds,
        const ShapeData shapeData,
        const GeometryData geometryData,
        const GeometrySdfData geometrySdfData,
        float4 *CR_RESTRICT nodeMomentumVelocityMass)
    {
        int nodeIdx = blockIdx.x * blockDim.x + threadIdx.x;
        if (nodeIdx >= numNodes)
            return;

        standardMpmUpdateGrid<true>(
            nodeIdx,
            gridBoundMin,
            cellSize,
            numNodesPerDim,
            integrationStepSize,
            nodeForce,
            numShapes,
            shapeIds,
            shapeData,
            geometryData,
            geometrySdfData,
            nodeMomentumVelocityMass);
    }

    CR_CUDA_GLOBAL void standardMpmGridToParticleKernel(
        const int numParticles,
        const Vec3f gridBoundMin,
        const Vec3f gridBoundMax,
        const float invCellSize,
        const Vec3i numNodesPerDim,
        const float integrationStepSize,
        const float4 *CR_RESTRICT nodeMomentumVelocityMass,
        const int numShapes,
        const int *CR_RESTRICT shapeIds,
        const ShapeData shapeData,
        const GeometryData geometryData,
        const GeometrySdfData geometrySdfData,
        float4 *CR_RESTRICT particlePositionMass,
        Vec3f *CR_RESTRICT particleVelocity,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn0,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn1,
        float4 *CR_RESTRICT particleGradientDeformationTensorColumn2)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numParticles)
            return;

        Mat3x3f particleGradientDeformationTensor(particleGradientDeformationTensorColumn0[idx],
                                                  particleGradientDeformationTensorColumn1[idx],
                                                  particleGradientDeformationTensorColumn2[idx]);

        standardMpmGridToParticle<true>(
            gridBoundMin,
            gridBoundMax,
            invCellSize,
            numNodesPerDim,
            integrationStepSize,
            nodeMomentumVelocityMass,
            numShapes,
            shapeIds,
            shapeData,
            geometryData,
            geometrySdfData,
            particlePositionMass[idx],
            particleVelocity[idx],
            particleGradientDeformationTensor);

        particleGradientDeformationTensorColumn0[idx] = particleGradientDeformationTensor.col0;
        particleGradientDeformationTensorColumn1[idx] = particleGradientDeformationTensor.col1;
        particleGradientDeformationTensorColumn2[idx] = particleGradientDeformationTensor.col2;
    }
} // namespace crmpm
